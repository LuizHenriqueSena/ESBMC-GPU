#include "hip/hip_runtime.h"
#include <call_kernel.h>
//xfail:BOOGIE_ERROR:data race
//--blockDim=2 --gridDim=1 --no-inline

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <sm_atomic_functions.h>

#define N 2

__global__ void race_test (unsigned int* i, int* A)
{
	int tid = threadIdx.x;
	int j = atomicAdd(i,tid);
	A[j] = tid;
}

int main(){

	unsigned int *i;
	int *A;
	unsigned int *dev_i;
	int *dev_A;

	A = (int*)malloc(N*sizeof(int));

	for (int t = 0; t < N; ++t){
		A[t] = 0;
		printf(" %d  ", A[t]);
	}

	i = (unsigned int*)malloc(sizeof(unsigned int));
	
	*i = 0;

	hipMalloc((void**)&dev_A, N*sizeof(int));
	hipMalloc((void**)&dev_i, sizeof(unsigned int));

	hipMemcpy(dev_A, A, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_i, i, sizeof(unsigned int), hipMemcpyHostToDevice);

		//race_test<<<1,N>>>(dev_i, dev_A);
		ESBMC_verify_kernel_u(race_test,1,N,dev_i,dev_A);
		
	hipMemcpy(A, dev_A, N*sizeof(int), hipMemcpyDeviceToHost);

/*	printf("\n");

	for (int t=0; t<N; t++){
		printf ("A[%d]=%d ", t, A[t]);
	}

	printf("\n\n");
*/	
	int tid = 0;
	for (int t=0; t<N;){
		printf ("A[%d]=%d ", t, A[t]);
		//assert(A[t] == tid); // A[t] == x , where x=[0,N]
		assert(A[t] == 0 || A[t] == 1); // A[t] == x , where x=[0,N-1]		
		tid++;
		t = t + tid;
	}

	free(A);
	free(i);
	hipFree(dev_A);
	hipFree(dev_i);

	return 0;
}
