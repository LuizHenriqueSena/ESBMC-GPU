#include "hip/hip_runtime.h"
//xfail:BOOGIE_ERROR
//error: possible null pointer access

#include <stdio.h>
#include <hipblas.h>
#include <hipDNN.h>

#define N 9//64



int main(){

	float* v;
	float* v2;	
	float* dev_v;
	float* dev_v2;
	float* dev_v3;
	float* dev_dlossdata;
	float* dev_result;
	float* dev_dbackpropagation;
	float* alpha;
	float* beta;
	float al = 1;
	float be = 0;
	//float* dev_result;
	
	//initializing cublas handle
	hipblasHandle_t cublasHandle;
	hipdnnHandle_t cudnnHandle;
	hipblasCreate(&cublasHandle);
	hipdnnCreate(&cudnnHandle);

	/* sets the size of v */
	v = (float*)malloc(N*sizeof(float));
	v2 = (float*)malloc(N*sizeof(float));

	for (int i = 0; i < N ; ++i){
		v[i] = i;
}	
	for (int i = 0; i < N; ++i) {
		v2[i] = i*i;
}
		
	hipMalloc((void**)&dev_v, N*sizeof(float)); /* visible only by GPU: __global__ functions */
	hipMalloc((void**)&dev_v2, N*sizeof(float));
	hipMalloc((void**)&dev_v3, N*sizeof(float));
	hipMalloc((void**)&dev_dlossdata, N*sizeof(float));
	hipMalloc((void**)&dev_result, N*sizeof(float));
	hipMalloc((void**)&dev_dbackpropagation, N*sizeof(float));
	hipMalloc((void**)&alpha, sizeof(float));
	hipMalloc((void**)&beta, sizeof(float));

	hipMemcpy(dev_v, v, N*sizeof(float), hipMemcpyHostToDevice);	
	hipMemcpy(dev_v2, v2, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(alpha, &al, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(beta, &be, sizeof(float), hipMemcpyHostToDevice);
	//This function operates the dev_v3= alpha*(dev_v) x dev_v2 + beta*dev_v3


	hipdnnActivationDescriptor_t activationDesc;
	hipdnnTensorDescriptor_t xDesc, yDesc, dxDesc, dyDesc;
	
	hipblasSgemm(cublasHandle,
			HIPBLAS_OP_N, HIPBLAS_OP_N,
			3, 1, 3,
			alpha,
			dev_v, 3,
			dev_v2, 3,
			beta,
			dev_v3, 3);

	hipdnnActivationForward(cudnnHandle, activationDesc, alpha, xDesc, dev_v3, beta, yDesc, dev_result);

	hipdnnActivationBackward(cudnnHandle, activationDesc, alpha, yDesc, dev_result, dyDesc, dev_dlossdata, xDesc, dev_v, beta, dxDesc, dev_dbackpropagation);
		//foor<<<1, N>>>(dev_v, N, c);
		//ESBMC_verify_kernel_fuintt(foor,1,N, dev_v, N, c);
		
	//hipMemcpy(v, dev_v, dimensionx*dimensiony*sizeof(float), hipMemcpyDeviceToHost);


	free(v);
	free(v2);
	hipFree(dev_v);
	hipFree(dev_v2);
	//hipFree(dev_result);

	return 0;
}
