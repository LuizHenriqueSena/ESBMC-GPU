#include "hip/hip_runtime.h"
//xfail:BOOGIE_ERROR
//error: possible null pointer access

#include <stdio.h>
#include <hipblas.h>

#define N 10//64



int main(){

	float* v;
	float* dev_v;
	int * dev_result;

	//initializing cublas handle
	hipblasHandle_t cublasHandle;
	hipblasCreate(&cublasHandle);

	/* sets the size of v */
	v = (float*)malloc(N*sizeof(float));

	for (int i = 0; i < N ; ++i){
		v[i] = i;
}	

	hipMalloc((void**)&dev_v, N*sizeof(float)); /* visible only by GPU: __global__ functions */
	hipMalloc((void**)&dev_result, sizeof(int));

	hipMemcpy(dev_v, v, N*sizeof(float), hipMemcpyHostToDevice);	

	//This function looks for the index of the smallest value of the vector dev_V
	hipblasIsamin(cublasHandle, N, dev_v, 1, dev_result);

		//foor<<<1, N>>>(dev_v, N, c);
		//ESBMC_verify_kernel_fuintt(foor,1,N, dev_v, N, c);
		
	//hipMemcpy(v, dev_v, dimensionx*dimensiony*sizeof(float), hipMemcpyDeviceToHost);


	free(v);
	hipFree(dev_v);

	return 0;
}
