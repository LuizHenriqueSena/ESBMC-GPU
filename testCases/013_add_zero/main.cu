#include "hip/hip_runtime.h"
#include <call_kernel.h>
//fail: data-race, all the threads write on A[0]

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <sm_atomic_functions.h>

#define N 2

__global__ void race_test (unsigned int* i, int* A)
{
	int tid = threadIdx.x;
	int j = atomicAdd(i,0);
  	A[j] = tid;
}

  int main(){

  	unsigned int *i;
  	int *A;
  	unsigned int *dev_i;
  	int *dev_A;

  	A = (int*)malloc(N*sizeof(int));

  	for (int t = 0; t < N; ++t){
  		A[t] = 11;
  		printf(" %d  ", A[t]);
  	}

  	i = (unsigned int*)malloc(sizeof(unsigned int));

	*i = 0;
  	
	hipMalloc((void**)&dev_A, N*sizeof(int));
  	hipMalloc((void**)&dev_i, sizeof(unsigned int));

  	hipMemcpy(dev_A, A, N*sizeof(int), hipMemcpyHostToDevice);
  	hipMemcpy(dev_i, i, sizeof(unsigned int), hipMemcpyHostToDevice);

  		//race_test<<<1,N>>>(dev_i, dev_A);
		ESBMC_verify_kernel_u(race_test,1,N,dev_i,dev_A);

  	hipMemcpy(A, dev_A, N*sizeof(int), hipMemcpyDeviceToHost);

  	for (int t=0; t<N;t++){
  		printf("A[%d]=%d; ", t, A[t]);
  	}
	
	//assert(A[0] == 11);
	assert(A[0] == 0 || A[0] == 1); // A[0] == i,where i = [0,N-1]

  	free(A);
  	free(i);
  	hipFree(dev_A);
  	hipFree(dev_i);

  	return 0;
}
