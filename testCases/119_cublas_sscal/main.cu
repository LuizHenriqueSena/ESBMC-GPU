#include "hip/hip_runtime.h"
//xfail:BOOGIE_ERROR
//error: possible null pointer access

#include <stdio.h>
#include <hipblas.h>

#define N 10//64



int main(){

	float* v;
	float* dev_v;
	float* alpha;
	float aux = 3;

	//initializing cublas handle
	hipblasHandle_t cublasHandle;
	hipblasCreate(&cublasHandle);

	/* sets the size of v */
	v = (float*)malloc(N*sizeof(float));

	for (int i = 0; i < N ; ++i){
		v[i] = i;
}	

	hipMalloc((void**)&dev_v, N*sizeof(float)); /* visible only by GPU: __global__ functions */
	hipMalloc((void**)&alpha, sizeof(float));

	hipMemcpy(dev_v, v, N*sizeof(float), hipMemcpyHostToDevice);	
	hipMemcpy(alpha, &aux, N*sizeof(float), hipMemcpyHostToDevice);
	//This function scales the vector dev_v by the factor alpha
	hipblasSscal(cublasHandle, N, alpha, dev_v, 1);

		//foor<<<1, N>>>(dev_v, N, c);
		//ESBMC_verify_kernel_fuintt(foor,1,N, dev_v, N, c);
		
	//hipMemcpy(v, dev_v, dimensionx*dimensiony*sizeof(float), hipMemcpyDeviceToHost);


	free(v);
	hipFree(dev_v);

	return 0;
}
