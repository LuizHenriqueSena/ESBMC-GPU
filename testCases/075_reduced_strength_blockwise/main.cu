#include "hip/hip_runtime.h"
//pass
//--blockDim=256 --gridDim=2 -DWIDTH=2064 --no-inline
#include <call_kernel.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

#define GRIDDIM 1
#define BLOCKDIM 2//256
#define WIDTH 2//2048
#define N WIDTH
/*
 * This kernel demonstrates a blockwise strength-reduction loop.
 * Each block is given a disjoint partition (of length WIDTH) of A.
 * Then each thread writes multiple elements in the partition.
 * It is not necessarily the case that WIDTH%blockDim.x == 0
 */

__global__ void k(int *A) {
//  __assert(blockDim.x <= WIDTH);
//#ifdef BLOCK_DIVIDES_WIDTH
//  //__assert(__mod_pow2(WIDTH, blockDim.x) == 0);
//#endif

  for (int i=threadIdx.x; i<WIDTH; i+=blockDim.x) {

//#ifndef BLOCK_DIVIDES_WIDTH
//    // working set(1) using global invariants
//    /*A*/__global_invariant(__write_implies(A, (blockIdx.x*WIDTH) <= __write_offset_bytes(A)/sizeof(int))),
//    /*B*/__global_invariant(__write_implies(A,                       __write_offset_bytes(A)/sizeof(int) < (blockIdx.x+1)*WIDTH)),
//    /*C*/__invariant(threadIdx.x <= i),
//    /*D*/__invariant(               i <= WIDTH+blockDim.x),
//         __invariant(i % blockDim.x == threadIdx.x),
//         __global_invariant(__write_implies(A, (((__write_offset_bytes(A)/sizeof(int)) % WIDTH) % blockDim.x) == threadIdx.x)),
//#else
//    // working set(2) iff WIDTH % blockDim.x == 0
//    /*A*/__invariant(__write_implies(A, (blockIdx.x*WIDTH) <= __write_offset_bytes(A)/sizeof(int))),
//    /*B*/__invariant(__write_implies(A,                       __write_offset_bytes(A)/sizeof(int) < (blockIdx.x+1)*WIDTH)),
//    /*C*/__invariant(threadIdx.x <= i),
//    /*D*/__invariant(               i <= WIDTH+blockDim.x),
//         __invariant(__uniform_int((i-threadIdx.x))),
//         __invariant(__uniform_bool(__enabled())),
//#endif

    A[blockIdx.x*WIDTH+i] = i;
  }

//#ifdef FORCE_FAIL
//  __assert(false);
//#endif
}

int main (){
	int *a;
	int *dev_a;
	int size = N*sizeof(int);

	hipMalloc((void**)&dev_a, size);

	a = (int*)malloc(size);

	for (int i = 0; i < N; i++)
		a[i] = 0;

//	printf("Old a:  ");
//	for (int i = 0; i < N; i++)
//		printf("%d	", a[i]);

	hipMemcpy(dev_a,a,size,hipMemcpyHostToDevice);

	//k <<<GRIDDIM, BLOCKDIM>>>(dev_a);
	ESBMC_verify_kernel(k,GRIDDIM,BLOCKDIM,dev_a);

	hipMemcpy(a,dev_a,size,hipMemcpyDeviceToHost);

//	printf("\nNew a:  ");

	for (int i = 0; i < N; i++){
//		printf("%d	", a[i]);
		assert(a[i]== i);
	}

	free(a);
	hipFree(dev_a);
	return 0;
}
