#include "hip/hip_runtime.h"
//xfail:BOOGIE_ERROR
//error: possible null pointer access

#include <stdio.h>
#include <hipblas.h>

#define N 10//64



int main(){

	float* v;
	float* v2;	
	float* dev_v;
	float* dev_v2;
	//float* dev_result;
	
	//initializing cublas handle
	hipblasHandle_t cublasHandle;
	hipblasCreate(&cublasHandle);

	/* sets the size of v */
	v = (float*)malloc(N*sizeof(float));
	v2 = (float*)malloc(N*sizeof(float));

	for (int i = 0; i < N ; ++i){
		v[i] = i;
}	
	for (int i = 0; i < N; ++i) {
		v2[i] = i*i;
}
		
	hipMalloc((void**)&dev_v, N*sizeof(float)); /* visible only by GPU: __global__ functions */
	hipMalloc((void**)&dev_v2, N*sizeof(float));
	//hipMalloc((void**)&dev_result, sizeof(float));

	hipMemcpy(dev_v, v, N*sizeof(float), hipMemcpyHostToDevice);	
	hipMemcpy(dev_v2, v2, N*sizeof(float), hipMemcpyHostToDevice);
	//This function copies the vector dev_v to dev_v2
	hipblasScopy(cublasHandle, N, dev_v, 1, dev_v2, 1);
		//foor<<<1, N>>>(dev_v, N, c);
		//ESBMC_verify_kernel_fuintt(foor,1,N, dev_v, N, c);
		
	//hipMemcpy(v, dev_v, dimensionx*dimensiony*sizeof(float), hipMemcpyDeviceToHost);


	free(v);
	free(v2);
	hipFree(dev_v);
	hipFree(dev_v2);
	//hipFree(dev_result);

	return 0;
}
