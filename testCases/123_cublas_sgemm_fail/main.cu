#include "hip/hip_runtime.h"
//xfail:BOOGIE_ERROR
//error: possible null pointer access

#include <stdio.h>
#include <hipblas.h>

#define N 9//64



int main(){

	float* v;
	float* v2;	
	float* dev_v;
	float* dev_v2;
	float* dev_v3;
	float* alpha;
	float* beta;
	float al = 1;
	float be = 0;
	//float* dev_result;
	
	//initializing cublas handle
	hipblasHandle_t cublasHandle;
	hipblasCreate(&cublasHandle);

	/* sets the size of v */
	v = (float*)malloc(N*sizeof(float));
	v2 = (float*)malloc(N*sizeof(float));

	for (int i = 0; i < N ; ++i){
		v[i] = i;
}	
	for (int i = 0; i < N; ++i) {
		v2[i] = i*i;
}
		
	hipMalloc((void**)&dev_v, N*sizeof(float)); /* visible only by GPU: __global__ functions */
	hipMalloc((void**)&dev_v2, N*sizeof(float));
	hipMalloc((void**)&dev_v3, N*sizeof(float));
	hipMalloc((void**)&alpha, sizeof(float));
	hipMalloc((void**)&beta, sizeof(float));

	hipMemcpy(dev_v, v, N*sizeof(float), hipMemcpyHostToDevice);	
	hipMemcpy(dev_v2, v2, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(alpha, &al, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(beta, &be, sizeof(float), hipMemcpyHostToDevice);
	//This function operates the dev_v3= alpha*(dev_v) x dev_v2 + beta*dev_v3

	hipblasSgemm(cublasHandle,
			HIPBLAS_OP_N, HIPBLAS_OP_N,
			3, 4, 3,
			alpha,
			dev_v, 3,
			dev_v2, 3,
			beta,
			dev_v3, 3);
		//foor<<<1, N>>>(dev_v, N, c);
		//ESBMC_verify_kernel_fuintt(foor,1,N, dev_v, N, c);
		
	//hipMemcpy(v, dev_v, dimensionx*dimensiony*sizeof(float), hipMemcpyDeviceToHost);


	free(v);
	free(v2);
	hipFree(dev_v);
	hipFree(dev_v2);
	//hipFree(dev_result);

	return 0;
}
