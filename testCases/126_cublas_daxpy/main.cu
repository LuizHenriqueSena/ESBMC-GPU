#include "hip/hip_runtime.h"
//xfail:BOOGIE_ERROR
//error: possible null pointer access

#include <stdio.h>
#include <hipblas.h>

#define N 9//64



int main(){

	double* v;
	double* v2;	
	double* dev_v;
	double* dev_v2;
	double* dev_v3;
	double* alpha;
	double* beta;
	double al = 1;
	double be = 0;
	//double* dev_result;
	
	//initializing cublas handle
	hipblasHandle_t cublasHandle;
	hipblasCreate(&cublasHandle);

	/* sets the size of v */
	v = (double*)malloc(N*sizeof(double));
	v2 = (double*)malloc(N*sizeof(double));

	for (int i = 0; i < N ; ++i){
		v[i] = i;
}	
	for (int i = 0; i < N; ++i) {
		v2[i] = i*i;
}
		
	hipMalloc((void**)&dev_v, N*sizeof(double)); /* visible only by GPU: __global__ functions */
	hipMalloc((void**)&dev_v2, N*sizeof(double));
	hipMalloc((void**)&dev_v3, N*sizeof(double));
	hipMalloc((void**)&alpha, sizeof(double));
	//hipMalloc((void**)&beta, sizeof(double));

	hipMemcpy(dev_v, v, N*sizeof(double), hipMemcpyHostToDevice);	
	hipMemcpy(dev_v2, v2, N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(alpha, &al, sizeof(double), hipMemcpyHostToDevice);
	//hipMemcpy(beta, &be, sizeof(double), hipMemcpyHostToDevice);
	//This function operates the dev_v3= alpha*(dev_v) x dev_v2 + beta*dev_v3

	hipblasDaxpy(cublasHandle, N,
                           alpha,
                           dev_v, 1,
                           dev_v2, 1);
		//foor<<<1, N>>>(dev_v, N, c);
		//ESBMC_verify_kernel_fuintt(foor,1,N, dev_v, N, c);
		
	//hipMemcpy(v, dev_v, dimensionx*dimensiony*sizeof(float), hipMemcpyDeviceToHost);


	free(v);
	free(v2);
	hipFree(dev_v);
	hipFree(dev_v2);
	//hipFree(dev_result);

	return 0;
}
