#include "hip/hip_runtime.h"
#include <call_kernel.h>
// Problem 1, Assignment #1
// count 6 using synchronization

#include <stdio.h>
#include <assert.h>

#define SIZE 256
#define BLOCKSIZE 32

__host__ void outer_compute(int *in_arr, int *out_arr);

int main(int argc, char **argv)
{
  int *in_array, *out_array;
  int sum=0, count=0;

  /* initialization */
  in_array = (int *) malloc(SIZE*sizeof(int));
  for (int i=0; i<SIZE; i++) {
    in_array[i] = rand()%10;
    printf("in_array[%d] = %d\n",i,in_array[i]);
    if (in_array[i] == 6)
    	count++;
  }
  out_array = (int *) malloc(BLOCKSIZE*sizeof(int));

  /* compute number of appearances of 6 */
  outer_compute(in_array, out_array);

  sum=out_array[0];

  printf ("The number 6 appears %d times in array of  %d numbers\n",sum,SIZE);
  assert(sum != count);
  getchar();
}

__device__ int compare(int a, int b) {
  if (a == b) return 1;
  return 0;
}

__global__ void compute(int *d_in,int *d_out) {
  int i;

  d_out[threadIdx.x] = 0;
  for (i=0; i<SIZE/BLOCKSIZE; i++) {
      d_out[threadIdx.x] += compare(d_in[i*BLOCKSIZE+threadIdx.x],6);
  }

  __syncthreads();
  /* Tree-structured results-gathing phase, for BLOCKSIZE=32 */
  if(threadIdx.x < 16){ d_out[threadIdx.x] += d_out[threadIdx.x + 16];} __syncthreads();
  if(threadIdx.x < 8) { d_out[threadIdx.x] += d_out[threadIdx.x + 8]; } __syncthreads();
  if(threadIdx.x < 4) { d_out[threadIdx.x] += d_out[threadIdx.x + 4]; } __syncthreads();
  if(threadIdx.x < 2) { d_out[threadIdx.x] += d_out[threadIdx.x + 2]; } __syncthreads();
  if(threadIdx.x < 1) { d_out[threadIdx.x] += d_out[threadIdx.x + 1]; }
}

__host__ void outer_compute(int *h_in_array, int *h_out_array) {
  int *d_in_array, *d_out_array;

  /* allocate memory for device copies, and copy input to device */
  hipMalloc((void **) &d_in_array,SIZE*sizeof(int));
  hipMalloc((void **) &d_out_array,BLOCKSIZE*sizeof(int));
  hipMemcpy(d_in_array,h_in_array,SIZE*sizeof(int),hipMemcpyHostToDevice);

  /* compute number of appearances of 8 for subset of data in each thread! */
  compute<<<1,BLOCKSIZE,(SIZE+BLOCKSIZE)*sizeof(int)>>>(d_in_array,d_out_array);

  hipMemcpy(h_out_array,d_out_array,BLOCKSIZE*sizeof(int),hipMemcpyDeviceToHost);
}

