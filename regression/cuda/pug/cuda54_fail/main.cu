#include "hip/hip_runtime.h"
#include <call_kernel.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>


#define SIZE 256
#define BLOCKSIZE 32

__host__ void outer_compute(int *in_arr, int *out_arr);

int main(int argc, char **argv)
{
  int *in_array, *out_array;
  int count;

  /* initialization */
  in_array = (int *) malloc(SIZE*sizeof(int));
  for (int i=0; i<SIZE; i++) {
    in_array[i] = rand()%10;
    printf("in_array[%d] = %d\n",i,in_array[i]);
    if (in_array[i] == 6)
    	count ++;
  }
  out_array = (int *) malloc(BLOCKSIZE*sizeof(int));

  /* compute number of appearances of 6 */
  outer_compute(in_array, out_array);

  /* out_array[0] contains result */
   printf ("The number 6 appears %d times in array of %d numbers\n",out_array[0],SIZE);
   assert(out_array[0] != count);
   getchar();

}

__device__ int compare(int a, int b) {
  if (a == b) return 1;
  return 0;
}

__global__ void compute(int *d_in,int *d_out, int *d_sum) {
  int i;

  d_out[threadIdx.x] = 0;
  for (i=0; i<SIZE/BLOCKSIZE; i++) {
    d_out[threadIdx.x] += compare(d_in[i*BLOCKSIZE+threadIdx.x],6);
  }

  for(i = 1; i <= BLOCKSIZE/2; i*=2) {
    __syncthreads();
    if(threadIdx.x % (i*2) == 0) {
      d_out[threadIdx.x] += d_out[threadIdx.x + i];
    }
  }
  if(threadIdx.x == 0) {
    *d_sum = d_out[0];
  }
}

__host__ void outer_compute(int *h_in_array, int *h_out_array) {
  int *d_in_array, *d_out_array, *d_sum;

  /* allocate memory for device copies, and copy input to device */
  hipMalloc((void **) &d_in_array,SIZE*sizeof(int));
  hipMalloc((void **) &d_out_array,BLOCKSIZE*sizeof(int));
  hipMalloc((void **) &d_sum,sizeof(int));
  hipMemcpy(d_in_array,h_in_array,SIZE*sizeof(int),hipMemcpyHostToDevice);

  /* compute number of appearances of 6 for subset of data in each thread! */
  compute<<<1,BLOCKSIZE,(SIZE+BLOCKSIZE)*sizeof(int)>>>(d_in_array,d_out_array,d_sum);

  /* Only need to copy d_sum into h_out_array[0], as it contains final answer */
  hipMemcpy(h_out_array,d_sum,sizeof(int),hipMemcpyDeviceToHost);
}

