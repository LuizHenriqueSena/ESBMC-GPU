#include "hip/hip_runtime.h"
#include <call_kernel.h>

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>

#define SIZE 256
#define BLOCKSIZE 32

__host__ void outer_compute(int *in_arr, int *out_arr);

int main(int argc, char **argv)
{
	int* in_array;
	int* sum = (int*)malloc(sizeof(int));
	*sum = 0;

	/* Initialization */
	in_array = (int*) malloc(SIZE * sizeof(int));
	for (int i = 0; i < SIZE; i++)
	{
		in_array[i] = rand() % 10;
		printf("in_array[%d] = %d\n", i, in_array[i]);
	}

	/* compute number of appearances of 6 */
	outer_compute(in_array, sum);


	printf ("The number 6 appears %d times in array of  %d numbers\n",*sum,SIZE);
	assert(*sum == 0);
	getchar();
}

__device__ int compare(int a, int b)
{
	if (a == b)
		return 1;
	return 0;
}

__global__ void compute(int *d_in,int *d_out, int* d_sum)
{
	int i;

	d_out[threadIdx.x] = 0;
	for (i = 0; i < SIZE / BLOCKSIZE; i++)
	{
		d_out[threadIdx.x] += compare(d_in[i * BLOCKSIZE + threadIdx.x], 6);
	}

	__syncthreads();

	for(int i = 2, j = 1; i <= BLOCKSIZE; i *= 2, j *= 2)
	{
		if(threadIdx.x % i == 0)
		{
			d_out[threadIdx.x] += d_out[threadIdx.x + j];
		}
		__syncthreads();
	}

	*d_sum = d_out[0];

}

__host__ void outer_compute(int *h_in_array, int *h_sum)
{
	int *d_in_array, *d_sum, *d_out_array;

	/* Allocate memory for device copies, and copy input to device */
	hipMalloc( (void **) &d_in_array,SIZE * sizeof(int) );
	hipMalloc( (void **) &d_sum, sizeof(int));
	hipMalloc( (void **) &d_out_array, BLOCKSIZE * sizeof(int) );
	hipMemcpy(d_in_array, h_in_array, SIZE * sizeof(int), hipMemcpyHostToDevice);

	/* Compute number of appearances of 6 for subset of data in each thread! */
	compute<<<1,BLOCKSIZE,(SIZE + BLOCKSIZE) * sizeof(int)>>>(d_in_array, d_out_array, d_sum);

	hipMemcpy(h_sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);
}

