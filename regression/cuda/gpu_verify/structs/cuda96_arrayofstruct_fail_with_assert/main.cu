#include "hip/hip_runtime.h"
#include <call_kernel.h>
//fail
//--gridDim=1 --blockDim=2 --no-inline

//This kernel is race-free.
//
//It uses uses struct-assignment, which is translated into a memcpy by clang and
//dealt with as a series of reads/writes by bugle.

#include <stdio.h>
#include <assert.h>

#define N 2

typedef struct {
  short x;
  short y;
} pair_t;

__global__ void k(pair_t *pairs) {
  pair_t fresh;
  fresh.x = 2; fresh.y = 3;
  pairs[threadIdx.x] = fresh;
}

int main(){
	pair_t *a;
	pair_t *dev_a;
	int size = N*sizeof(pair_t);

	a = (pair_t*)malloc(size);

	/* initialization of a */
	a[0].x = 5; a[1].x = 6;
	a[0].y = 5; a[1].y = 6;

	hipMalloc((void**)&dev_a, size);

	hipMemcpy(dev_a,a,size, hipMemcpyHostToDevice);

	printf("a:\n");
	for (int i = 0; i < N; i++)
		printf("a[%d].x : %d  \ta[%d].y : %d\n", i, a[i].x, i, a[i].y);

	k<<<1,N>>>(dev_a);

	hipMemcpy(a,dev_a,size,hipMemcpyDeviceToHost);

	printf("new a:\n");
	for (int i = 0; i < N; i++) {
		printf("a[%d].x : %d  \ta[%d].y : %d\n", i, a[i].x, i, a[i].y);
		assert(!(a[i].x == 2  && a[i].y == 3));
	}
	
	free(a);
	
	hipFree(&dev_a);

	return 0;
}
