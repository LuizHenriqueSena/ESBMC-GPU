#include "hip/hip_runtime.h"
#include <call_kernel.h>
//xfail:BOOGIE_ERROR
//--gridDim=1 --blockDim=2 --no-inline

//This kernel has a null pointer access.

#include <stdio.h>
#define memcpy(dst, src, len) __builtin_memcpy(dst, src, len)
#define N 2

typedef struct {
  short x;
  short y;
} s_t; //< sizeof(s_t) == 4

__global__ void k(s_t *in, s_t *out) {
	memcpy(&out[threadIdx.x], 0, sizeof(s_t));
}

int main(){
	s_t *a;
	s_t *dev_a;
	s_t *c;
	s_t *dev_c;
	int size = N*sizeof(s_t);

	a = (s_t*)malloc(size);
	c = (s_t*)malloc(size);

	/* initialization of a (the in) */
	a[0].x = 5; a[0].y = 6;
	a[1].x = 5; a[1].y = 6;

	/* initialization of c (the out) */
	c[0].x = 2; c[0].y = 3;
	c[1].x = 2; c[1].y = 3;

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_c, size);

	hipMemcpy(dev_a,a,size, hipMemcpyHostToDevice);
	hipMemcpy(dev_c,c,size, hipMemcpyHostToDevice);

	printf("a:\n");
	for (int i = 0; i < N; i++)
		printf("a[%d].x : %d  \ta[%d].y : %d\n", i, a[i].x, i, a[i].y);

	printf("c:\n");
	for (int i = 0; i < N; i++)
		printf("c[%d].x : %d  \tc[%d].y : %d\n", i, c[i].x, i, c[i].y);

	k<<<1,N>>>(dev_a, dev_c);

	hipMemcpy(c,dev_c,size,hipMemcpyDeviceToHost);

	printf("new c:\n");
	for (int i = 0; i < N; i++)
		printf("c[%d].x : %d  \tc[%d].y : %d\n", i, c[i].x, i, c[i].y);

	hipFree(&dev_a);
	hipFree(&dev_c);

	return 0;
}
