#include "hip/hip_runtime.h"
#include <call_kernel.h>
//pass
//--gridDim=1 --blockDim=2 --no-inline

//This kernel is race-free.
//
//It uses uses memcpy and copies fewer bytes than the struct size so we have to
//handle the arrays in and out at the byte-level.
#include <stdio.h>
#include <assert.h>

#define memcpy(dst, src, len) __builtin_memcpy(dst, src, len)
#define N 2

typedef struct {
  short x;
  short y;
  char z;
} s_t; //< sizeof(s_t) == 6

__global__ void k(s_t *in, s_t *out) {
  size_t len = 5;
  memcpy(&out[threadIdx.x], &in[threadIdx.x], len);
}

int main(){
	s_t *a;
	s_t *dev_a;
	s_t *c;
	s_t *dev_c;
	int size = N*sizeof(s_t);

	a = (s_t*)malloc(size);
	c = (s_t*)malloc(size);

	/* initialization of a (the in) */
	a[0].x = 5; a[0].y = 6; a[0].z = 'i';
	a[1].x = 5; a[1].y = 6; a[1].z = 'i';

	/* initialization of c (the out) */
	c[0].x = 2; c[0].y = 3; c[0].z = 'o';
	c[1].x = 2; c[1].y = 3; c[1].z = 'o';

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_c, size);

	hipMemcpy(dev_a,a,size, hipMemcpyHostToDevice);
	hipMemcpy(dev_c,c,size, hipMemcpyHostToDevice);

	printf("a:\n");
	for (int i = 0; i < N; i++)
		printf("a[%d].x : %d  \ta[%d].y : %d\ta[%d].z : %c\n", i, a[i].x, i, a[i].y, i, a[i].z);

	printf("c:\n");
	for (int i = 0; i < N; i++)
		printf("c[%d].x : %d  \tc[%d].y : %d\tc[%d].z : %c\n", i, c[i].x, i, c[i].y, i, c[i].z);

	k<<<1,N>>>(dev_a, dev_c);

	hipMemcpy(c,dev_c,size,hipMemcpyDeviceToHost);

	printf("new c:\n");
	for (int i = 0; i < N; i++) {
		printf("c[%d].x : %d  \tc[%d].y : %d\tc[%d].z : %c\n", i, c[i].x, i, c[i].y, i, c[i].z);
		assert (!(c[i].x = 5 && c[i].y = 6 && c[i].z = 'i'));
	}
	
	free(a); free(c);
	hipFree(&dev_a);
	hipFree(&dev_c);

	return 0;
}
