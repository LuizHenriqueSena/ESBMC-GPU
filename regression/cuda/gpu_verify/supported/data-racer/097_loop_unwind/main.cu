#include "hip/hip_runtime.h"
//xfail:BOOGIE_ERROR
//--blockDim=512 --gridDim=64 --loop-unwind=2 --no-inline
//kernel.cu: error: possible write-write race on B

#include <call_kernel.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <assert.h>

#define N 2//512

extern "C" {

__global__ void helloCUDA(float *A)
{
    __shared__ float B[256];

    for(int i = 0; i < N*2; i ++) {
        B[i] = A[i];
    }
}

}

int main() {

	float *A;
	float *dev_A;

	float size= N*sizeof(float);

	A=(float*)malloc(size);

	hipMalloc((void**)&dev_A, size);

	for (int i = 0; i < N; i++)
		A[i] = 5;


	hipMemcpy(dev_A, A, size, hipMemcpyHostToDevice);
	
//		helloCUDA<<<64,N>>>(dev_A);
	ESBMC_verify_kernel(helloCUDA, 1, N, dev_A);

	hipMemcpy(A, dev_A, size, hipMemcpyDeviceToHost);

	hipFree(dev_A);
	free(A);

}
