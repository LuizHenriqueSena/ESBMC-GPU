#include "hip/hip_runtime.h"
//xfail:BOOGIE_ERROR
//--blockDim=64 --gridDim=64 --no-inline
//
#include "hip/hip_runtime.h"
#define N dim*dim
#define dim 2

__global__ void foo() {

  __shared__ int a;

  a = threadIdx.x;
}

int main(){

	foo <<<N,N>>> ();
	//ESBMC_verify_kernel(foo, dim,dim);

	hipDeviceSynchronize();

}
