//pass
//--blockDim=1024 --gridDim=1024 --no-inline

#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math_functions.h>

#define DIM 2 //1024 in the future
#define N 2//DIM*DIM

__global__ void mul24_test (int* A, int* B)
{
  int idxa          = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
  unsigned int idxb = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;

  A[idxa] = idxa;
  B[idxb] = idxa;
}

int main (){
	int *a, *b;
	int *dev_a, *dev_b;
	int size = N*sizeof(int);

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);

	a = (int*)malloc(size);
	b = (int*)malloc(size);

	for (int i = 0; i < N; i++)
		a[i] = 1;

	for (int i = 0; i < N; i++)
		b[i] = 1;

	hipMemcpy(dev_a,a,size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,size, hipMemcpyHostToDevice);

	mul24_test<<<DIM,DIM>>>(dev_a,dev_b);
	//ESBMC_verify_kernel(mul24_test,1,N,dev_a,dev_b);

	hipMemcpy(a,dev_a,size,hipMemcpyDeviceToHost);
	hipMemcpy(b,dev_b,size,hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++)
		assert (a[i] == i);	

	for (int i = 0; i < N; i++) 
		assert (b[i] == i);	

	free(a); free(b);

	hipFree(dev_a);
	hipFree(dev_b);

	return 0;
}
