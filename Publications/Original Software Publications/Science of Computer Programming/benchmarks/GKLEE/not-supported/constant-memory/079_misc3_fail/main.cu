#include "hip/hip_runtime.h"
//xfail:BOOGIE_ERROR
//possible attempt to modify constant memory
//You can modify the values of the constants, uncomment the lines 14 and 16 to analyze this case.

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <assert.h>

#define N 2//1024

__constant__ int A[N] = {0, 1, 2, 3};

__global__ void foo(int *B) {

//	assert(A[0]==0);
	A[threadIdx.x] = B[threadIdx.x];
//	assert(A[0]==0); // the constant memory was modified!!!
	__syncthreads();

	B[threadIdx.x] = A[threadIdx.x];

}

int main(){

	int *a;
	int *dev_a;
	int size = N*sizeof(int);

	hipMalloc((void**)&dev_a, size);

	a = (int*)malloc(size);

	for (int i = 0; i < N; i++)
		a[i] = 1;

	hipMemcpy(dev_a,a,size, hipMemcpyHostToDevice);

	printf("a:  ");
	for (int i = 0; i < N; i++)
		printf("%d	", a[i]);

	foo<<<1,N>>>(dev_a);
	//ESBMC_verify_kernel(foo, 1, N, dev_a);

	hipMemcpy(a,dev_a,size,hipMemcpyDeviceToHost);

	printf("\nThe function results:\n   ");

	for (int i = 0; i < N; i++){
		printf("%d	", a[i]);
	//		assert(a[i]==i);
	}

	free(a);

	hipFree(dev_a);

	return 0;
}
