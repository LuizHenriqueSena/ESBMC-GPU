#include "hip/hip_runtime.h"
//pass
//--blockDim=1024 --gridDim=1 --no-inline

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <sm_atomic_functions.h>
#include <stdio.h>

#define N 1 //1024

__global__ void definitions (int* A, unsigned int* B, unsigned long long int* C, float* D)
{
	assert(*D == 0.0);
	//assert(*D == 0.0 || *D == 5);
	assert(*D == 5.0);
/**/	atomicAdd(A,10);
	atomicSub(A,10);
	atomicExch(A,10);
	atomicMin(A,10);
	atomicMax(A,10);
	atomicAnd(A,10);
	atomicOr(A,10);
	atomicXor(A,10);
  	atomicCAS(A,10,11);

/**/	atomicAdd(B,10);
	atomicSub(B,10);
	atomicExch(B,10);
	atomicMin(B,10);
	atomicMax(B,10);
	atomicAnd(B,10);
	atomicOr(B,10);
	atomicXor(B,10);
	atomicInc(B,10);
	atomicDec(B,10);
  	atomicCAS(B,10,11);

/**/	atomicAdd(C,10);
	atomicExch(C,10);
	atomicMin(C,10);
	atomicMax(C,10);
	atomicAnd(C,10);
	atomicOr(C,10);
	atomicXor(C,10);
  	atomicCAS(C,10,11);

	atomicAdd(D,10.0);
	atomicExch(D,10.0);
}

int main (){

	int a = 5;
	int *dev_a;

	hipMalloc ((void**) &dev_a, sizeof(int));

	hipMemcpy(dev_a, &a, sizeof(int),hipMemcpyHostToDevice);

	unsigned int b = 5;
	unsigned int *dev_b;

	hipMalloc ((void**) &dev_b, sizeof(unsigned int));

	hipMemcpy(dev_b, &b, sizeof(unsigned int),hipMemcpyHostToDevice);

	unsigned long long int c = 5;
	unsigned long long int *dev_c;

	hipMalloc ((void**) &dev_c, sizeof(unsigned long long int));

	hipMemcpy(dev_c, &c, sizeof(unsigned long long int),hipMemcpyHostToDevice);
/**/
	float d = 5;
	assert(d==5);
	float *dev_d;// = (float*)malloc (sizeof(float));
	dev_d = (float*)malloc (sizeof(float));

	//hipMalloc ((void**) &dev_d, sizeof(float));

//	hipMemcpy(dev_d, &d, sizeof(float),hipMemcpyHostToDevice);
	memcpy(dev_d,&d, sizeof(float));

	assert (*dev_d == 0);
		definitions <<<1,N>>>(dev_a,dev_b,dev_c,dev_d);
		//ESBMC_verify_kernel_four(definitions,1,N,dev_a,dev_b,dev_c,dev_d);
		
/**/	hipMemcpy(&a,dev_a,sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(&b,dev_b,sizeof(unsigned int),hipMemcpyDeviceToHost);
	hipMemcpy(&c,dev_c,sizeof(unsigned long long int),hipMemcpyDeviceToHost);
	hipMemcpy(&d,dev_d,sizeof(float),hipMemcpyDeviceToHost);

/**/	printf("A: %d\n", a); assert(a == 0 || a == 11);
	printf("B: %u\n", b); assert(b == 0 || b == 11);
	printf("C: %u\n", c); assert(c == 0 || c == 11);
	//printf("D: %f\n", d); assert(d == 10.0f || d == 15.0f || d == 5.0f || d == 35.0f);

/**/	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipFree(dev_d);
	return 0;
}
