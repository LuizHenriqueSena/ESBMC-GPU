#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_ 
#define _KERNEL_H_

#include "config.h"
#include "my_cutil.h"

__global__ void kernel(int *a, int *b, int *c){
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

#endif
