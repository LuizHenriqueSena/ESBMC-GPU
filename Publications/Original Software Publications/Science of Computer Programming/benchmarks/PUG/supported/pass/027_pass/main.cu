#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_ 
#define _KERNEL_H_

#include "config.h"
#include "my_cutil.h"

__global__ void kernel(int *a, int *b, int *c){
	c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

#endif
