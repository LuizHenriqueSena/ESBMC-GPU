#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_ 
#define _KERNEL_H_

#include "config.h"
#include "my_cutil.h"

__global__ void kernel(void *data) {
	unsigned int *idata = (unsigned int*) data;
	idata[threadIdx.x] = bitreverse1(idata[threadIdx.x]);
}

#endif
