#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_ 
#define _KERNEL_H_

#include "config.h"
#include "my_cutil.h"

__global__ void kernel(float *v, unsigned int size, unsigned int i)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    funcType f;

    if (i == 1)
      f = multiplyByTwo;
    else if (i == 2)
      f = divideByTwo;
    else
      f = NULL;

    if (tid < size)
    {
        float x = (*f)(v, tid);
        x += multiplyByTwo(v, tid);
        v[tid] = x;
    }
}

#endif
