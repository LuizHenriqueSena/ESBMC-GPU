#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_ 
#define _KERNEL_H_

#include "config.h"
#include "my_cutil.h"

__global__ void kernel(int* A) {

   A[ blockIdx.x*blockDim.x + threadIdx.x ] += (A[ (blockIdx.x + 1)*blockDim.x + threadIdx.x ]);

}

#endif
