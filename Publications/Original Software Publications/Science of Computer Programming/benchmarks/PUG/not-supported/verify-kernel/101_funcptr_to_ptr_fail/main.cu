#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_ 
#define _KERNEL_H_

#include "config.h"
#include "my_cutil.h"

__global__ void kernel(float *v, funcType f, unsigned int size, int i)
{
	assert(i != 0);

	uint tid = blockIdx.x * blockDim.x + threadIdx.x;

    void *x = (void*)f;	/*ptr_to_ptr*/
    
    if (i == 0)				//*the null pointer occurs when i ==0, this is the case*//
		x = x + 5;

    funcType g = (funcType)x;

    if (tid < size)
    {
        v[tid] = (*g)(v, tid);
    }
}

#endif
