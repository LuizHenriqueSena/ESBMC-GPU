#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_ 
#define _KERNEL_H_

#include "config.h"
#include "my_cutil.h"

__global__ void kernel(hiprandState *state, float *A) {
   A[threadIdx.x] =  hiprand(&state[threadIdx.x]); // the pseudo random number returned by 'hiprand' is an unsigned int
}

#endif
