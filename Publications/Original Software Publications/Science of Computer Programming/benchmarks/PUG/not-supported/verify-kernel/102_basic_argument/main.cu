#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_ 
#define _KERNEL_H_

#include "config.h"
#include "my_cutil.h"

__global__ void kernel(float *v, funcType* f, unsigned int size)
{

	/************************************************************/
	assert(*f == divideByTwo || *f == multiplybyTwo);
	/************************************************************/

	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        v[tid] = (*f)(v, tid);
    }
}

#endif
