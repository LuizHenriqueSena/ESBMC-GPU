#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_ 
#define _KERNEL_H_

#include "config.h"
#include "my_cutil.h"

__global__ void kernel(hiprandStateMtgp32_t *state, float *A) {

	A[threadIdx.x] = hiprand(&state[threadIdx.x]);
}

#endif
