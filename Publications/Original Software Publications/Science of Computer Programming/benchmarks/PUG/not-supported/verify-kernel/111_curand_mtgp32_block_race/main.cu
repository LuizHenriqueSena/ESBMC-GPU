#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_ 
#define _KERNEL_H_

#include "config.h"
#include "my_cutil.h"

__global__ void kernel(hiprandStateMtgp32_t *state, float *A) {
  if (threadIdx.x == 0) {
    A[blockIdx.x] = hiprand(state);
  }
}

#endif
