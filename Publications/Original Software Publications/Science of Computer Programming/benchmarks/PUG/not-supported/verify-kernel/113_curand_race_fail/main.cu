#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_ 
#define _KERNEL_H_

#include "config.h"
#include "my_cutil.h"

__global__ void kernel(hiprandState *state, float *A) { // test: replace hiprandState for hiprandStateXORWOW_t
   A[threadIdx.x] = hiprand_uniform(state);
}

#endif
