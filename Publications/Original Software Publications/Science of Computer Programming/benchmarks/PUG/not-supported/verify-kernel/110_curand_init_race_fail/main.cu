#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_ 
#define _KERNEL_H_

#include "config.h"
#include "my_cutil.h"

__global__ void kernel(hiprandState *state, unsigned int *A) {
   hiprand_init(0, 0, 0, state);

   __syncthreads();

   A[threadIdx.x] =  hiprand(&state[threadIdx.x]);
//   if (threadIdx.x == 0) {
  //   A[0] = hiprand(state);
   //}
}

#endif
