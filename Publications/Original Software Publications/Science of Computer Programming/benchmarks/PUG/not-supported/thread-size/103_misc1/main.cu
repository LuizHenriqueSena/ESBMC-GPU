#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_ 
#define _KERNEL_H_

#include "config.h"
#include "my_cutil.h"

__global__ void kernel(int x)
{
    __shared__ float S[256*32];
    __shared__ float F[256];

    unsigned int idx;

    //initialise data on shared memory
    for(int i = 0;
   //         __invariant(__implies(__write(S), ((__write_offset_bytes(S)/sizeof(float)) % blockDim.x) == threadIdx.x)),
            i < x;
            i += (blockDim.x/32)) /* translate: i = 0; i < 143; i+=16 , total de iterações: 8*/
		
    {
        if((i+(threadIdx.x/32)) < x){
            idx = (i+(threadIdx.x/32))*32+(threadIdx.x%32);
            S[idx] = F[i+(threadIdx.x/32)];
        }
    }

}

#endif
