#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_ 
#define _KERNEL_H_

#include "config.h"
#include "my_cutil.h"

__global__ void kernel(int *c, const int *a){
    int col = (blockDim.x * blockIdx.x) + threadIdx.x;
	int row = (blockDim.y * blockIdx.y) + threadIdx.y;
    c[index(row,col,4)] = a[index(col, row, 4)] ;
}

#endif
