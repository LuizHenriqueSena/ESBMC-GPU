#include "hip/hip_runtime.h"
#ifndef _KERNEL_H_ 
#define _KERNEL_H_

#include "config.h"
#include "my_cutil.h"

__global__ void kernel (int* A)
{
	int tid = threadIdx.x;
	int warp = tid / 2;//32;
	int* B = A + (warp*2);//32);
	A[tid] = B[(tid + 1)%2];//32];
}

#endif
