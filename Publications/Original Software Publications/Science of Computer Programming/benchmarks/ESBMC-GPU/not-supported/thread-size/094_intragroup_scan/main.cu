#include "hip/hip_runtime.h"
#include <call_kernel.h>
//pass
//--blockDim=512 --gridDim=1 --warp-sync=32 --no-inline

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

#define N 32//128//512

__device__ static __attribute__((always_inline)) void scan_warp (int* A)
{
	unsigned int tid = threadIdx.x;
	unsigned int lane = tid % 32;

	if (lane >= 1) A[tid] = A[tid - 1] + A[tid]; // this conditional is always true!!
	if (lane >= 2) A[tid] = A[tid - 2] + A[tid];
	if (lane >= 4) A[tid] = A[tid - 4] + A[tid];
	if (lane >= 8) A[tid] = A[tid - 8] + A[tid];
	if (lane >= 16) A[tid] = A[tid - 16] + A[tid];
}

__global__ void scan (int* A)
{
	unsigned int tid = threadIdx.x;
	unsigned int lane = tid % 32;

	int temp [32] = {0};
	scan_warp(A);
	__syncthreads();

	if (lane == 31)	// ?????????
		temp[tid / 32] = A[tid];
	__syncthreads();

	if (tid / 32 == 0)
		scan_warp(temp);
	__syncthreads();

	A[tid] += temp[tid/32];

}


int main(){
	int *a;
	int *dev_a;
	int size = N*sizeof(int);

	hipMalloc((void**)&dev_a, size);

	a = (int*)malloc(size);

	for (int i = 0; i < N; i++)
		a[i] = i;

	hipMemcpy(dev_a,a,size, hipMemcpyHostToDevice);

	printf("old a:  ");
	for (int i = 0; i < N; i++)
		printf("%d	", a[i]);

	//scan<<<1,N>>>(dev_a);
	ESBMC_verify_kernel(scan, 1, N, dev_a);

	hipMemcpy(a,dev_a,size,hipMemcpyDeviceToHost);

	printf("\nnew a:  ");
	for (int i = 0; i < N; i++)
		printf("%d	", a[i]);

	free(a);

	hipFree(dev_a);

	return 0;
}
