#include "hip/hip_runtime.h"
#include <call_kernel.h>
//xfail:BOOGIE_ERROR
//--blockDim=1024 --gridDim=1
//null pointer access
// ALTOUGH, IT WORKS

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> 
#define N 2//4//8

__global__ void foo(int *H) {
  size_t tmp = (size_t)H; //type cast
  tmp += sizeof(int);
  int *G = (int *)tmp;
  G -= 1;					//POSSIBLE NULL POINTER ACCESS
  G[threadIdx.x] = threadIdx.x;
  __syncthreads();
  H[threadIdx.x] = G[threadIdx.x];
}

int main() {

	int *a;
	int *dev_a;
	int size = N*sizeof(int);

	hipMalloc((void**)&dev_a, size);

	a = (int*)malloc(N*size);

	for (int i = 0; i < N; i++)
		a[i] = 1;

	hipMemcpy(dev_a,a,size, hipMemcpyHostToDevice);

	printf("a:  ");
	for (int i = 0; i < N; i++)
		printf("%d	", a[i]);

	//foo<<<1,N>>>(dev_a);
	ESBMC_verify_kernel(foo, 1, N, dev_a);

	hipMemcpy(a,dev_a,size,hipMemcpyDeviceToHost);

	printf("\nFunction Results:\n   ");

	for (int i = 0; i < N; i++)
		printf("%d	", a[i]);

	free(a);

	hipFree(dev_a);

	return 0;
}

