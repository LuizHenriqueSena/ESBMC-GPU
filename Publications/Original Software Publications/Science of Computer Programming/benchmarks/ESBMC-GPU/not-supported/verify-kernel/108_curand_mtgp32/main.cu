#include "hip/hip_runtime.h"
#include <call_kernel.h>
//pass
//--blockDim=256 --gridDim=1 --no-inline

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
//#include <hiprand.h>

#define N 2 //256

__global__ void curand_test(hiprandStateMtgp32_t *state, float *A) {

	A[threadIdx.x] = hiprand(&state[threadIdx.x]);
}

int main() {
	hiprandStateMtgp32_t tipo; // Mtgp32_t
	float *a;
	float *dev_a;
	tipo *dev_state;
	mtgp32_kernel_params *devKernelParams;

	int size = N*sizeof(float);

	a = (float*)malloc(size);
	hipMalloc ((void**) &dev_a, size);

	printf("old a:  ");
	for (int i = 0; i < N; i++)
	printf("%f	", a[i]);

	hipMalloc ( (void**) &dev_state, N*sizeof( tipo ) );

	hipMalloc((void**)&devKernelParams,sizeof(mtgp32_kernel_params));

	hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams); /* Set up constant parameters for the mtgp32 generator */
		/* mtgp32dc_params_fast_11213 is a constant of the type mtgp32_params_fast, it is a system constant */
		/* devKernelParams is the destination*/

	hiprandMakeMTGP32KernelState(dev_state, mtgp32dc_params_fast_11213, devKernelParams,N, 1234); /* Set up initial states for the mtgp32 generator */
		/*
		 * \param s - pointer to an array of states in device memory
		 * \param params - Pointer to an array of type mtgp32_params_fast_t in host memory
		 * \param k - pointer to a structure of type mtgp32_kernel_params_t in device memory
		 * \param n - number of parameter sets/states to initialize
		 * \param seed - seed value
		 *
		 * */

//	curand_test<<<1,N>>>(dev_state, dev_a);
	ESBMC_verify_kernel(curand_test,1,N,dev_state,dev_a);	

	hipMemcpy(a,dev_a,size,hipMemcpyDeviceToHost);

	printf("\nnew a:  ");
	for (int i = 0; i < N; i++)
		printf("%f	", a[i]);

	free(a);
	hipFree(&dev_a);
	hipFree(&dev_state);
	hipFree(&devKernelParams);

	return 0;
}
