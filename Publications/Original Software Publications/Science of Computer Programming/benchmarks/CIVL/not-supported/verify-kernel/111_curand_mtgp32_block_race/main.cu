#include "hip/hip_runtime.h"
//fail
//--blockDim=256 --gridDim=1 --no-inline

#include <hiprand/hiprand_kernel.h>
//#include <hiprand/hiprand_mtgp32_host.h>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>


#define N 32 //16

__global__ void curand_test(hiprandStateMtgp32_t *state, float *A) {
  if (threadIdx.x == 0) {
    A[blockIdx.x] = hiprand(state);
  }
}

int main() {
	typedef hiprandStateMtgp32_t tipo;
	float *a;
	float *dev_a;
	tipo *dev_state;
	mtgp32_kernel_params *devKernelParams;

	int size = N*sizeof(float);

	a = (float*)malloc(size);
	hipMalloc ((void**) &dev_a, size);

	printf("old a:  ");
	for (int i = 0; i < N; i++)
	printf("%f	", a[i]);

	hipMalloc ( (void**) &dev_state, N*sizeof( tipo ) );

	hipMalloc((void**)&devKernelParams,sizeof(mtgp32_kernel_params));

	hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams); /* Set up constant parameters for the mtgp32 generator */
		/* mtgp32dc_params_fast_11213 is a constant of the type mtgp32_params_fast, it is a system constant */
		/* devKernelParams is the destination*/

	hiprandMakeMTGP32KernelState(dev_state, mtgp32dc_params_fast_11213, devKernelParams,N, 3412); /* Set up initial states for the mtgp32 generator */
		/*
		 * \param s - pointer to an array of states in device memory
		 * \param params - Pointer to an array of type mtgp32_params_fast_t in host memory
		 * \param k - pointer to a structure of type mtgp32_kernel_params_t in device memory
		 * \param n - number of parameter sets/states to initialize
		 * \param seed - seed value
		 *
		 * */

	curand_test<<<1,N>>>(dev_state, dev_a);
	//ESBMC_verify_kernel(curand_test,1,N,dev_state,dev_a);

	hipMemcpy(a,dev_a,size,hipMemcpyDeviceToHost);

	printf("\nnew a:  ");
	for (int i = 0; i < N; i++)
		printf("%f	", a[i]);

	free(a);
	hipFree(&dev_a);
	hipFree(&dev_state);
	hipFree(&devKernelParams);

	return 0;
}
