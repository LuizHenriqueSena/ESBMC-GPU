//xfail:BOOGIE_ERROR
//--blockDim=1024 --gridDim=1 --no-inline
//error: possible null pointer access

#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define N 2//8

#define tid (blockIdx.x * blockDim.x + threadIdx.x)

__device__ float multiplyByTwo(float *v, unsigned int index)
{
    return v[index] * 2.0f;
}

__device__ float divideByTwo(float *v, unsigned int index)
{
    return v[index] * 0.5f;
}

typedef float(*funcType)(float*, unsigned int);

__global__ void foo(float *v)
{
    funcType f = (funcType)3; // it's a null pointer
    v[tid] = (*f)(v, tid);
}

int main(){

	float* w;
	float* dev_w;

	int size = N*sizeof(float);

	w =(float*) malloc(size);

	for (int i = 0; i < N; ++i){
		w[i] = i;
	}


	hipMalloc((void**)&dev_w, size);

	hipMemcpy(dev_w,w, size,hipMemcpyHostToDevice);

	foo <<<1,N>>>(dev_w);
	//ESBMC_verify_kernel_f(foo, 1, N, dev_w);

	hipMemcpy(w,dev_w,size,hipMemcpyDeviceToHost);

	printf("\nw:");
	for (int i = 0; i < N; ++i){
		printf(" %f	",	w[i]);
//		assert(!(w[i] == i));
	}

	//printf ("\n (float) functype: %f", divideByTwo)//3.5;

	free(w);
	hipFree(dev_w);

	return 0;
}
