//xfail:BUGLE_ERROR
//--gridDim=1 --blockDim=32 --no-inline

//This kernel is not-racy: memset is called with variable value.

//#define memset(dst,val,len) __builtin_memset(dst,val,len)

#define N 2//32

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <assert.h>
#include <stdlib.h>

__device__ int bar(void){
	int value;
	return value;
}

__global__ void kernel(uint4 *out) {
  uint4 vector;
  int val = bar();
   memset(&vector, val, 16);
  out[threadIdx.x] = vector;
  /**/
}

int main(){
	uint4 *a;
	uint4 *dev_a;
	int size = N*sizeof(uint4);

	a = (uint4*)malloc(size);

	/* initialization of a */
	for (int i = 0; i < N; i++) {
		a[i].x = i; a[i].y = i; a[i].z = i, a[i].w = i;
	}

	hipMalloc((void**)&dev_a, size);

	hipMemcpy(dev_a,a,size, hipMemcpyHostToDevice);

/*	printf("a:\n");
	for (int i = 0; i < N; i++)
		printf("a[%d].x : %d  \ta[%d].y : %d\ta[%d].z : %d\ta[%d].w : %d\n", i, a[i].x, i, a[i].y, i, a[i].z, i, a[i].w);
*/
		kernel<<<1,N>>>(dev_a);
		//ESBMC_verify_kernel_u(kernel,1,N,dev_a);

	hipMemcpy(a,dev_a,size,hipMemcpyDeviceToHost);

	printf("new a:\n");
	for (int i = 0; i < N; i++) {
/*		printf("a[%d].x : %d  \ta[%d].y : %d\ta[%d].z : %d\ta[%d].w : %d\n", i, a[i].x, i, a[i].y, i, a[i].z, i, a[i].w);
*/		assert(a[i].x == 0);			
		assert(a[i].y == 0);
		assert(a[i].z == 0);			
		assert(a[i].w == 0);
}

	hipFree(dev_a);
	free(a);
	return 0;
}
