#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math_functions.h>

#define N 2//64

__global__ void foo(float *x, float y) {
	x[threadIdx.x] = __exp10f(y);	// pow(10,y), in this  case pow(10,2) = 100
}

int main(void){
	float *A;
	float *dev_A;

	float size= N*sizeof(float);

	A=(float*)malloc(size);

	hipMalloc((void**)&dev_A, size);

	hipMemcpy(dev_A, A, size, hipMemcpyHostToDevice);

	foo<<<1,N>>>(dev_A, 2);
	//ESBMC_verify_kernel_f(foo, 1, N, dev_A, 2);

	hipMemcpy(A, dev_A, size, hipMemcpyDeviceToHost);

	printf("\n");

	for(int t=0; t<N; t++){
		printf("%.1f ", A[t]);
		assert (A[t] != 100);
	}

	hipFree(dev_A);
	free(A); 

	return 0;
}
