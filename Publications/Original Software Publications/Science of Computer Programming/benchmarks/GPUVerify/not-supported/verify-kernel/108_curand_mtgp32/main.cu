//pass
//--blockDim=256 --gridDim=1 --no-inline

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>
//#include <curand.h>

#define N 2 //256

__global__ void curand_test(hiprandStateMtgp32_t *state, float *A) {

	A[threadIdx.x] = hiprand(&state[threadIdx.x]);
}

