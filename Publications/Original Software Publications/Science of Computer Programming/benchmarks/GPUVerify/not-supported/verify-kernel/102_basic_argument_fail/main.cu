#include "hip/hip_runtime.h"
//fail: o argumento não é passado com sucesso
//--blockDim=1024 --gridDim=1 --no-inline
#include <stdio.h>
#include <stdlib.h>

#define N 8

__device__ float multiplyByTwo(float *v, unsigned int tid) {

    return v[tid] * 2.0f;
}

__device__ float divideByTwo(float *v, unsigned int tid) {

    return v[tid] * 0.5f;
}

typedef float(*funcType)(float*, unsigned int);

__global__ void foo(float *v, funcType* f, unsigned int size)
{
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	__assert(*f == divideByTwo || *f == multiplybyTwo);

    if (tid < size) {
        v[tid] = (*f)(v, tid);
    }
}

