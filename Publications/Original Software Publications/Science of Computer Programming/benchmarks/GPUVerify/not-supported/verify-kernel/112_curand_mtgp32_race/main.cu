//pass
//--blockDim=256 --gridDim=1 --no-inline

#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
//#include <curand_precalc.h>
//#include <curand_mtgp32_host.h>
#include <stdio.h>

#define N 256

__global__ void curand_test(hiprandStateMtgp32_t *state, float *A) {

	A[threadIdx.x] = hiprand(&state[threadIdx.x]);
}

