//xfail:BOOGIE_ERROR
//--blockDim=2 --gridDim=1 --no-inline
//Write by thread .+kernel\.cu:8:21:

#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <assert.h>

#define N 4

__global__ void curand_test(hiprandState *state, float *A) { // test: replace curandState for curandStateXORWOW_t
   A[threadIdx.x] = hiprand_uniform(state);
}

