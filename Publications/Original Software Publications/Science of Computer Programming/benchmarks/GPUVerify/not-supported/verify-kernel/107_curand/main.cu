#include "hip/hip_runtime.h"
//pass
//--blockDim=512 --gridDim=1 --no-inline

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>


#define N 2 //512

__global__ void curand_test(hiprandState *state, float *A) {
   A[threadIdx.x] =  hiprand(&state[threadIdx.x]); // the pseudo random number returned by 'hiprand' is an unsigned int
}

