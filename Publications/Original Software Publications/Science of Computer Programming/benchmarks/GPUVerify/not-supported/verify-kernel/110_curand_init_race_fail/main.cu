//xfail:BOOGIE_ERROR
//--blockDim=2 --gridDim=1 --no-inline
//Write by thread .+kernel.cu:8:4:
// to threadIdx.x != 0 we have 'data race'.

#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

#define N 8 //2

__global__ void init_test(hiprandState *state, unsigned int *A) {
   hiprand_init(0, 0, 0, state);

   __syncthreads();

   A[threadIdx.x] =  hiprand(&state[threadIdx.x]);
//   if (threadIdx.x == 0) {
  //   A[0] = curand(state);
   //}
}

