//xfail:BUGLE_ERROR
//--gridDim=1 --blockDim=32 --no-inline

//This kernel is racy: memset is called with variable length.
//#define memset(dst,val,len) __builtin_memset(dst,val,len)

#define N 2//32

#include <stdio.h>
#include <hip/hip_runtime.h>


__device__ int bar(void){
	int value;
	return value;
}

__global__ void kernel(uint4 *out) {
  uint4 vector;
  int len = bar();
   memset(&vector, 5, len); /*modify manually the value of len to see the bugs*/
  out[threadIdx.x] = vector;
}
