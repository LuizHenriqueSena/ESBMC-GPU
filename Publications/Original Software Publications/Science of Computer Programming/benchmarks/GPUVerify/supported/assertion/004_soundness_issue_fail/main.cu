#include "hip/hip_runtime.h"
//xfail:BOOGIE_ERROR
//--blockDim=128 --gridDim=16 --no-inline
//assert\(false\)

#include <stdio.h>
#include <assert.h>
#include "hip/hip_runtime.h"



typedef void(*funcType)(float*);

__device__ void a(float *v)
{
	printf ("funcA with p%f = %f", *v, *v);
}
__device__ void b(float *v)
{
	printf ("funcB with p%f = %f", *v, *v);
}

__device__ void c(float *v)
{
	printf ("funcC with p%f = %f", *v, *v);
}

__device__ void d(float *v)
{
	printf ("funcD with p%f = %f", *v, *v);
}

__device__ void e(float *v)
{
	printf ("funcE with p%f = %f", *v, *v);
}

__global__ void should_fail(float * __restrict p1, float * __restrict p2, float * __restrict p3, float * __restrict p4, float * __restrict p5, int x, int y)
{
	__requires(x == 4);
	__requires(y == 4);
	funcType fp = a;

    switch(x) {
    case 1:
        fp = &a;
        break;
    case 2:
        fp = &b;
        break;
    case 3:
        fp = &c;
        break;
    case 4:
        fp = &d;
        break;
    default:
        fp = &e;
        break;
    }

    switch(y) {
    case 1:
        fp(p1);
        break;
    case 2:
        fp(p2);
        break;
    case 3:
        fp(p3);
        break;
    case 4:
        fp(p4);
        break;
    default:
        fp(p5);
        break;
    }

   __assert(0);
}

