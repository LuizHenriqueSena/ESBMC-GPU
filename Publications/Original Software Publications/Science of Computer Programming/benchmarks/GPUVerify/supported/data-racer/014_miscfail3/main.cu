//xfail:BOOGIE_ERROR
//main.cu: error: possible read-write race
//however, this didn't happen in the tests
//altough in CUDA providing the inline keyword should still keep a copy of the function around,
//this kind of access is considered a error by ESBMC
//ps: the values from A[N-1-offset) to A[N-1] always will receive unpredictable values,
//because they acess values because they access memory positions that were not initiated

#include <stdio.h>
#include <hip/hip_runtime.h>

#define tid threadIdx.x
#define N 2//1024

__device__ inline void inlined(int *A, int offset)
{
   int temp = A[tid + offset];
   A[tid] += temp;
}

__global__ void inline_test(int *A, int offset) {
  inlined(A, offset);
}

