//xfail:BOOGIE_ERROR
//--warp-sync=32 --blockDim=32 --gridDim=1 --equality-abstraction --no-inline
//kernel.cu:10

#include <hip/hip_runtime.h>

#include <stdio.h>
#include <assert.h>
#define N 2//32

__global__ void foo(int * A) {
    A[0] = 1;
    A[1] = 1;
    A[threadIdx.x] = 0;
//__assert(A[0] == 1 | A[1] == 1 | A[2] == 1);
}

