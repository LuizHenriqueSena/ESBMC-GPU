//pass
//--blockDim=1024 --gridDim=1 --no-inline

#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 2 //1024

__global__ void definitions (int* A, unsigned int* B, unsigned long long int* C)
{
  atomicOr(A,10);

  atomicOr(B,7);

  atomicOr(C,5);
}

