/******************************************* ternarytest.cu **************************************/
/*mostra 0 no índice 0, "c" no índice 1 e nos índice pares, mostra valor lixo nos demais índices */

#include <stdio.h>
#include "hip/hip_runtime.h"

#include <assert.h>

#define N 2//64

__global__ void foo(float* A, float c) {

		A[threadIdx.x == 0 ? 1 : 2*threadIdx.x] = c;

}

