#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <assert.h>
//#include <time.h>
#define N 2//(64*64)//(2048*2048)
#define THREADS_PER_BLOCK 2//512

__global__ void Asum(int *a, int *b, int *c){
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	c[index] = a[index] + b[index];
}

