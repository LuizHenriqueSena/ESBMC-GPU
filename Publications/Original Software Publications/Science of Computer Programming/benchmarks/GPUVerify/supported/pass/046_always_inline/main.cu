#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#define N 2//64

__device__ int bar() __attribute__((always_inline));

__device__ int bar()
{
  return 5;
}

__global__ void foo()
{
  int x = bar();
  __assert(x == 5);
//  printf("%d ", x);

}

